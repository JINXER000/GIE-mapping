#include "hip/hip_runtime.h"
#include "hokuyo_helper.h"
#include "hokuyo_interfaces.h"
#include "par_wave/voxmap_utils.cuh"

namespace HOKUYO_FAST
{


__global__
void setLocalOccupancy(LocMap loc_map,
                      SCAN_DEPTH_TPYE *detph_data,
                      Projection proj,
                      ScanParam param,
                      int3* VB_keys_loc_D,
                      bool for_motion_planner,
                      int rbt_r2_grids)
{
    int3 local_crd, glb_crd;
    local_crd.z = blockIdx.x;
    local_crd.y = threadIdx.x;

    float idea_depth, real_depth;
    int theta_idx;
    float3 glb_pos;

    for (local_crd.x = 0; local_crd.x < loc_map._local_size.x; ++local_crd.x)
    {
        int idx_1d=loc_map.coord2idx_local(local_crd);
        glb_crd = loc_map.loc2glb(local_crd);

        // set the self pose as known
        if(for_motion_planner)
        {
            int3 crd2center = local_crd -loc_map._half_shift;
            if(crd2center.x*crd2center.x + crd2center.y*crd2center.y+crd2center.z*crd2center.z <= rbt_r2_grids)
            {
                loc_map.set_vox_type(local_crd,VOXTYPE_FREE);
                VB_keys_loc_D[idx_1d] = get_VB_key(glb_crd);
                continue;
            }
        }
        glb_pos=loc_map.coord2pos(glb_crd);

        SCAN_HELPER::G2L(glb_pos,proj,param,loc_map._voxel_width,theta_idx,idea_depth);

        if (idea_depth < 0 || theta_idx<0 || theta_idx>=param.scan_num)
        {
            VB_keys_loc_D[idx_1d] = EMPTY_KEY;
            continue;
        }


        real_depth=detph_data[theta_idx];

        if (isnan(real_depth) || real_depth <= 0.3f)
        {
            VB_keys_loc_D[idx_1d] = EMPTY_KEY;
            continue;
        }


        if (idea_depth < real_depth - 0.3f)
        {
            loc_map.set_vox_type(local_crd,VOXTYPE_FREE);
            VB_keys_loc_D[idx_1d] = get_VB_key(glb_crd);
        }
        else if (idea_depth > real_depth + 0.3)
        {
            // Not observable
            VB_keys_loc_D[idx_1d] = EMPTY_KEY; // vox type is unknonw
        }
        else if(glb_pos.z >= loc_map._update_min_h && glb_pos.z <= loc_map._update_max_h)
        {
            loc_map.set_vox_type(local_crd,VOXTYPE_OCCUPIED);
            VB_keys_loc_D[idx_1d] = get_VB_key(glb_crd);
        }else
        {
            VB_keys_loc_D[idx_1d] = EMPTY_KEY;
        }
    }
}

void localOGMKernels(LocMap* loc_map, SCAN_DEPTH_TPYE *detph_data, Projection proj, ScanParam param,
                        int3* VB_keys_loc_D, bool for_motion_planner, int rbt_r2_grids)
{

    const int gridSize = loc_map->_local_size.z;
    const int blkSize = loc_map->_local_size.y;
    setLocalOccupancy<<<gridSize,blkSize>>>(*loc_map,detph_data,proj,param,VB_keys_loc_D,
                                           for_motion_planner,rbt_r2_grids);
}
}

