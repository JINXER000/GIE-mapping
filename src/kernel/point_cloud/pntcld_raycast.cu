#include "hip/hip_runtime.h"
#include "pntcld_interfaces.h"
#include <float.h>
#include "ray_cast.h"
#include "par_wave/voxmap_utils.cuh"

namespace PNTCLD_RAYCAST
{

__device__ __forceinline__
bool clearRayLoc(LocMap &loc_map,const int3 &crd, const float &val1, const float &val2, const int &time)
{
    if (loc_map.get_vox_type(crd) != VOXTYPE_OCCUPIED)
    {
        loc_map.atom_add_type_count(crd, -1);
        return true;
    }
    return false;
}


__global__
void getAllocKeys(LocMap loc_map, int3* VB_keys_loc_D, bool for_motion_planner, int rbt_r2_grids)
{
    // get the z and y coordinate of the grid we are about to scan
    int3 loc_crd;
    loc_crd.z = blockIdx.x;
    loc_crd.y = threadIdx.x;

    for (loc_crd.x = 0; loc_crd.x < loc_map._local_size.x; ++loc_crd.x)
    {

        // set grids around as known and free
        if (for_motion_planner)
        {
            int3 crd2center = loc_crd -loc_map._half_shift;
            if(crd2center.x*crd2center.x + crd2center.y*crd2center.y+crd2center.z*crd2center.z <= rbt_r2_grids)
            {
                loc_map.set_vox_count(loc_crd, -1);
            }

        }

        int idx_1d=loc_map.coord2idx_local(loc_crd);

        int count = loc_map.get_vox_count(loc_crd);

        if (count == 0)
        {
            VB_keys_loc_D[idx_1d] = EMPTY_KEY; // vox type is unknown
        }else
        {
            if(count>0)
            {
                loc_map.set_vox_type(loc_crd,VOXTYPE_OCCUPIED);
            }else
            {
                loc_map.set_vox_type(loc_crd,VOXTYPE_FREE);
            }
            int3 glb_crd = loc_map.loc2glb(loc_crd);
            VB_keys_loc_D[idx_1d] = get_VB_key(glb_crd);
        }
    }
}



__global__
void freeLocObs(LocMap loc_map, float3 *pnt_cld, Projection proj, int pnt_sz, int time)
{
    int ring_id = blockIdx.x;
    int scan_id = threadIdx.x;
    int id = threadIdx.x + blockIdx.x *blockDim.x;

    if(id >= pnt_sz)
        return;

    float3 glb_pos = proj.L2G*pnt_cld[id];

    RAY::rayCastLoc(loc_map, proj.origin,  glb_pos, time, 0.707f*loc_map._local_size.x*loc_map._voxel_width, &clearRayLoc);
}


__global__
void registerLocObs(LocMap loc_map, float3 *pnt_cld, Projection proj,  int pnt_sz, int time,
                    int ext_free_num, float3* freeBBX_ll, float3* freeBBX_ur)
{
    int ring_id = blockIdx.x;
    int scan_id = threadIdx.x;
    int id = threadIdx.x + blockIdx.x *blockDim.x;

    if(id >= pnt_sz)
        return;

    float3 glb_pos = proj.L2G*pnt_cld[id];
    bool within_height_limit = glb_pos.z >= loc_map._update_min_h && glb_pos.z <= loc_map._update_max_h;
    bool inside_clear_AABB = false;

    // clean for collision map
    for  (int i=0; i< ext_free_num; i++)
    {
        if(insideAABB( glb_pos, freeBBX_ll[i], freeBBX_ur[i]))
        {
            inside_clear_AABB = true;
            break;
        }
    }

    if (within_height_limit && !inside_clear_AABB)
    {
        int3 glb_crd = loc_map.pos2coord(glb_pos);
        int3 loc_crd = loc_map.glb2loc(glb_crd);

        loc_map.set_vox_type(loc_crd,VOXTYPE_OCCUPIED);
        loc_map.atom_add_type_count(loc_crd,1);
    }
}


void localOGMKernels(LocMap* loc_map, float3 *pnt_cld, Projection proj, PntcldParam param,
                     int3* VB_keys_loc_D, int time, bool for_motion_planner, int rbt_r2_grids,
                     Ext_Obs_Wrapper* ext_obsv)
{
    // Register the point clouds
    registerLocObs<<<param.valid_pnt_count/256+1, 256>>>(*loc_map,pnt_cld,proj,param.valid_pnt_count,time,
                                                         ext_obsv->ext_free_num,
                                                         raw_pointer_cast(&(ext_obsv->freeBBX_ll_D[0])),
                                                         raw_pointer_cast(&(ext_obsv->freeBBX_ur_D[0])));

    // Free the empty areas
    freeLocObs<<<param.valid_pnt_count/256+1, 256>>>(*loc_map,pnt_cld,proj,param.valid_pnt_count,time);

    const int gridSize = loc_map->_local_size.z;
    const int blkSize = loc_map->_local_size.y;
    getAllocKeys<<<gridSize,blkSize>>>(*loc_map,VB_keys_loc_D, for_motion_planner, rbt_r2_grids);

}
}
