#include "hip/hip_runtime.h"
#include "camera_helper.h"
#include "realsense_interfaces.h"
#include "par_wave/voxmap_utils.cuh"

namespace REALSENSE_FAST
{


__global__
void setLocalOccupancy(LocMap loc_map,
                      REALSENSE_DEPTH_TPYE *d_depth,
                      Projection proj,
                      CamParam param,
                       int3* VB_keys_loc_D,
                       bool for_motion_planner,
                       int rbt_r2_grids)
{
    int3 local_crd;
    float idea_depth;
    int2 pix;

    local_crd.z = blockIdx.x;
    local_crd.y = threadIdx.x;

    for (local_crd.x = 0; local_crd.x < loc_map._local_size.x; ++local_crd.x)
    {

        int idx_1d=loc_map.coord2idx_local(local_crd);

        int3 glb_crd= loc_map.loc2glb(local_crd);

        // set the self pose as known
        if(for_motion_planner)
        {
            int3 crd2center = local_crd -loc_map._half_shift;
            if(crd2center.x*crd2center.x + crd2center.y*crd2center.y+crd2center.z*crd2center.z <= rbt_r2_grids)
            {
                loc_map.set_vox_type(local_crd,VOXTYPE_FREE);
                VB_keys_loc_D[idx_1d] = get_VB_key(glb_crd);
                continue;
            }
        }

        float3 glb_pos=loc_map.coord2pos(glb_crd);
        CAM_HELPER::G2L(glb_pos,proj,param,pix,idea_depth);

        if (idea_depth <= 0.3f || idea_depth >6.0f ||
            pix.x < 0 || pix.x >= param.cols ||
            pix.y < 0 || pix.y >= param.rows)
        {
            VB_keys_loc_D[idx_1d] = EMPTY_KEY;
            continue;
        }

        float real_depth=d_depth[param.cols*pix.y+pix.x];

        if (real_depth <= 0.21f)
        {
            VB_keys_loc_D[idx_1d] = EMPTY_KEY;
            continue;
        }


        if (isnan(real_depth))
        {
            if(param.valid_NaN)
                real_depth = SENS_FAR_DIST;
            else
            {
                VB_keys_loc_D[idx_1d] = EMPTY_KEY;
                continue;
            }
        }

        if (idea_depth < real_depth - loc_map._voxel_width)
        {
            loc_map.set_vox_type(local_crd,VOXTYPE_FREE);
            VB_keys_loc_D[idx_1d] = get_VB_key(glb_crd);
        }
        else if (idea_depth > real_depth + loc_map._voxel_width)
        {
            // Not observable
            VB_keys_loc_D[idx_1d] = EMPTY_KEY; // vox type is unknonw
        }
        else if(glb_pos.z >= loc_map._update_min_h && glb_pos.z <= loc_map._update_max_h)
        {
            loc_map.set_vox_type(local_crd,VOXTYPE_OCCUPIED);
            VB_keys_loc_D[idx_1d] = get_VB_key(glb_crd);
        }else
        {
            VB_keys_loc_D[idx_1d] = EMPTY_KEY;
        }
    }
}


void localOGMKernels(LocMap* loc_map, REALSENSE_DEPTH_TPYE *detph_data, Projection proj, CamParam param,
                     int3* VB_keys_loc_D, bool for_motion_planner, int rbt_r2_grids)
{
    const int gridSize = loc_map->_local_size.z;
    const int blkSize = loc_map->_local_size.y;
    setLocalOccupancy<<<gridSize,blkSize>>>(*loc_map, detph_data, proj,param,VB_keys_loc_D,
                                            for_motion_planner,rbt_r2_grids);
}
}
